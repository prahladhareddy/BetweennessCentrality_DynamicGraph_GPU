#include "_graph.h"
#include "hip/hip_runtime.h"
#include "general.cuh"

int* _copy( int * cpu_arr ,int n)
{
    int * arr;
    hipMalloc(&arr,(n)*sizeof(int));
    hipMemcpy(arr, cpu_arr, sizeof(int) * (n), hipMemcpyHostToDevice);
    return arr;
}

struct cuda_copies copy_to_gpu(graph* g)
{
    struct cuda_copies cpy;
    int n = g->N;
    int m = g->M;
    cpy.offset_arr = _copy(g->offset_arr,n+1);
    cpy.edge_arr = _copy(g->edge_arr , m);
    cpy.cut_ver = _copy(g->cut_ver,n);
    cpy.bcc_arr = _copy(g->bcc_arr,m);
    cpy.subg_size = _copy(g->subg_size,m);
    return cpy;
}

void _free (struct cuda_copies c)
{
    hipFree(c.offset_arr);
    hipFree(c.edge_arr);
    hipFree(c.cut_ver);
    hipFree(c.subg_size);
    hipFree(c.bcc_arr);
}

// __device__ void  init_vec(struct vec* v)
// {
//     v->size=0;
//     v->max_size=4;
//     v->arr = (int*) malloc (4*sizeof(int));
// }

// __device__ void  push(struct vec *v1,int a)
// {
//     int s_max = v1->max_size;
//     int* arr = v1->arr;
//     int s = v1->size;
//     if(s<s_max)
//     {
//         arr[s]=a;
//         v1->size++;
//     }
//     else 
//     {
//         int* arr2 = (int*) malloc (2*s_max*sizeof(int));
//         for(int i=0;i<s;i++)
//         {
//             arr2[i]=arr[i];
//         }
//         free(arr);
//         arr2[s]=a;
//         v1->arr = arr2;
//         v1->size++;
//         v1->max_size = 2*s_max;
//     }
// }