#include "hip/hip_runtime.h"
#include "_graph.h"
#include "hip/hip_runtime.h"
#include "general.cuh"

__global__ void k_find_btw(int* offset_arr ,int* edge_arr , double* btw ,int* edge_rev_arr,int N,int M,int off,double *g_ans,int *g_dis,int *g_path,int *g_stack,int *g_queue,int* g_parent,int* g_vis)
{
    
    int id = blockIdx.x +  off;
    if(id>=N)
    {
        return;
    }
    if(threadIdx.x == 0 && id<N)
    {
        double *ans = &g_ans[N*id];
        int *vis = &g_vis[N*id];
        int *dis = &g_dis[N*id];
        int *path = &g_path[N*id];
        int* stack = &g_stack[N*id];
        int* queue = &g_queue[N*id];
        int* parent = &g_parent[M*id];
        for(int i=0;i<N;i++)
        {
            vis[i]=0;
        }
        for(int i=0;i<N;i++)
        {
            ans[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            dis[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            path[i]=0;   
        }

        for(int i=0;i<M;i++)
        {
            parent[i]=0;
        }

        queue[0]=id;
        stack[0]=id;
        int q_start=0;
        int q_end=1;

        int s_top=1;

        vis[id]=1;
        while(q_start<q_end)
        {
            int t = queue[q_start];
            q_start++;

            for(int i=offset_arr[t];i<offset_arr[t+1];i++)
            {
                int x = edge_arr[i];
                if(vis[x]==0)
                {
                    path[x]+=path[t];
                    if(t==id)
                    {
                        path[x]=1;
                    }
                    queue[q_end]=x;
                    q_end++;
                    vis[x]=1;
                    stack[s_top]=x;
                    s_top++;
                    parent[edge_rev_arr[i]]=1;
                    dis[x]=dis[t]+1;
                }
                else if((dis[t]+1)==dis[x])
                {
                    path[x]+=path[t];
                    parent[edge_rev_arr[i]]=1;
                }
            }
        }

        while(s_top!=0)
        {
            int t = stack[s_top-1];
            s_top--;
            for(int i=offset_arr[t];i<offset_arr[t+1];i++)
            {
                if(parent[i]==1)
                {
                    int x = edge_arr[i];
                    double div = (double)path[x]/(double)path[t];
                    ans[x]+=1.0*(1.0+ans[t]*1.0)*(1.0*div);
                }
            }
        }
        for(int i=0;i<N;i++)
        {
            btw[id*N+i]=ans[i];
        }
    }
}



void graph::find_btw()
{
    btw = (double*) malloc (N*sizeof(double));
    int* gpu_offset_arr;
    int* gpu_edge_arr;
    double* gpu_btw;
    int* gpu_edge_rev_arr;
    double* result = (double*) malloc (N*N*sizeof(double));
    hipMalloc( &gpu_offset_arr, sizeof(int) * (1+N) );
    hipMalloc( &gpu_edge_arr, sizeof(int) * (M) );
    hipMalloc( &gpu_btw, sizeof(double) * (N*N) );
    hipMalloc( &gpu_edge_rev_arr, sizeof(int) * (M) );
    hipMemcpy(gpu_offset_arr, offset_arr, sizeof(int) * (1+N), hipMemcpyHostToDevice);
    hipMemcpy(gpu_edge_arr, edge_arr, sizeof(int) * M, hipMemcpyHostToDevice);
    hipMemcpy(gpu_edge_rev_arr, edge_rev_arr, sizeof(int) * M, hipMemcpyHostToDevice);

    double *g_ans ;
    int *g_vis ;
    int *g_dis ;
    int *g_path ;
    int* g_stack ;
    int* g_queue ;
    int* g_parent ;

    hipMalloc(&g_ans,(N*N*sizeof(double)));
    hipMalloc(&g_dis,(N*N*sizeof(int)));
    hipMalloc(&g_path,(N*N*sizeof(int)));
    hipMalloc(&g_stack,(N*N*sizeof(int)));
    hipMalloc(&g_queue,(N*N*sizeof(int)));
    hipMalloc(&g_vis,(N*N*sizeof(int)));
    hipMalloc(&g_parent,(N*M*sizeof(int)));


    k_find_btw<<<N,1>>> (gpu_offset_arr,gpu_edge_arr,gpu_btw,gpu_edge_rev_arr,N,M,0,g_ans,g_dis,g_path,g_stack,g_queue,g_parent,g_vis);

    hipMemcpy(result,gpu_btw,N*N*sizeof(double),hipMemcpyDeviceToHost);

    hipFree(g_ans);
    hipFree(g_parent);
    hipFree(g_dis);
    hipFree(g_path);
    hipFree(g_stack);
    hipFree(g_queue);
    hipFree(g_vis);

    for(int i=0;i<N;i++)
    {
        btw[i]=0;
    }
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
        {
            btw[i]+=result[j*N+i];
        }
        btw[i]/=2;
        cout<<btw[i]<<" ";
    }
    cout<<"\n";
}


void __global__  k_btw_bcc( struct cuda_copies c,int N,int M,int *arr,int comp,int* lock,int factor,double* bc)
{
    int id = blockIdx.x;
    if(threadIdx.x==0)
    {
        double *ans1 = (double *)malloc(N*sizeof(double));
        double *ans2 = (double *)malloc(N*sizeof(double));
        int *vis = (int *)malloc(N*sizeof(int));
        int *dis = (int *)malloc(N*sizeof(int));
        int *path = (int *)malloc(N*sizeof(int));

        int* stack = (int*)malloc(N*sizeof(int));
        int* queue = (int*)malloc(N*sizeof(int));
        struct vec* parent = (struct vec*)malloc(N*sizeof(struct vec));

        for(int i=0;i<N;i++)
        {
            init_vec(&parent[i]);
        }
        
        for(int i=0;i<N;i++)
        {
            vis[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            ans1[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            ans2[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            dis[i]=0;
        }

        for(int i=0;i<N;i++)
        {
            path[i]=0;   
        }

        queue[0]=arr[id];
        stack[0]=arr[id];
        int q_start=0;
        int q_end=1;

        int s_top=1;

        vis[arr[id]]=1;
        while(q_start<q_end)
        {
            int t = queue[q_start];
            q_start++;

            for(int i=c.offset_arr[t];i<c.offset_arr[t+1];i++)
            {
                if(c.bcc_arr[i]!=comp)
                {
                    continue;
                }

                int x = c.edge_arr[i];
                if(vis[x]==0)
                {
                    path[x]+=path[t];
                    if(t==arr[id])
                    {
                        path[x]=1;
                    }
                    queue[q_end]=x;
                    q_end++;
                    vis[x]=1;
                    stack[s_top]=x;
                    s_top++;
                    push(&parent[x],t);
                    dis[x]=dis[t]+1;
                }
                else if((dis[t]+1)==dis[x])
                {
                    path[x]+=path[t];
                    push(&parent[x],t);
                }
            }
        }

        int f1 = c.cut_ver[arr[id]];
        int f2=0;
        int s1 = 0;
        int s2 = 0;
        if(f1)
        {
            for(int i= c.offset_arr[arr[id]];i<c.offset_arr[arr[id]+1];i++)
            {
                if(c.bcc_arr[i]!=comp)
                {
                    s1+=c.subg_size[i];
                }
            }
        }
        
        while(s_top!=0)
        {
            int t = stack[s_top-1];
            s_top--;
            f2 = c.cut_ver[t];
            s2=0;
            if(f2)
            {
                for(int i= c.offset_arr[t];i<c.offset_arr[t+1];i++)
                {
                    if(c.bcc_arr[i]!=comp)
                    {
                        s2+=c.subg_size[i];
                    }
                }
            }
            if(f1&&f2 && t!=arr[id])
            {
                ans1[t]=s1*s2;
            }
            else if(f1&&f2)
            {
                ans1[t]=s1*(N-s1-1);
            }
            for(int i=0;i<parent[t].size;i++)
            {
                int p = parent[t].arr[i];
                ans2[p]+= ((double)path[p]/(double)path[t])*(1.0+ans2[t]*1.0);
                if(f1)
                {
                    ans1[p]+=ans1[t]*1.0*(double)path[p]/(double)path[t];
                }
            }

            double a=0;
            if(t!=arr[id])
            {
                a+=factor*ans2[t]/2.0;
            }
            if(f1)
            {
                a+=factor*(ans2[t]*s1)*1.0;
                a+=factor*ans1[t]/2.0;

            }
            bool leaveLoop = false;
            while (!leaveLoop) {
                if (atomicExch(lock, 1) == 0) {
                    
                    bc[t]+= a;
                    leaveLoop = true;
                    atomicExch(lock,0);
                }
            } 

        }

    }
}

void up_comp(graph*g,int comp,struct cuda_copies c,int flag,double* gpu_btw)
{   
    int num_vert = g->bcc_verts[comp].size();
    int* arr = (int *)malloc(num_vert*sizeof(int));
    int ind=0;
    for(auto iter = (g->bcc_verts[comp].begin());iter != (g->bcc_verts[comp].end());iter++)
    {
        arr[ind] = *iter;
        ind++;
    }
    
    int* gpu_arr = _copy(arr,num_vert);
    int x=0;
    int* lock = _copy(&x,1);
    k_btw_bcc<<<num_vert,1>>>(c,g->N,g->M,gpu_arr,comp,lock,flag,gpu_btw);
    hipDeviceSynchronize();
}

void update_btw(graph** g,vector<edge> added,vector<edge> deleted)
{
    graph* g1 =*g;
    vector<edge> new_list = up(g1->edgeList,added,deleted);
    int m = g1->M+added.size()-deleted.size();
    graph* g2 = new graph(g1->N,m,new_list);
    g2->do_basic();
    vector<vector<int>>aff_comp = affected_comps (g1,g2);

    double* gpu_btw;
    hipMalloc(&gpu_btw,g1->N*sizeof(double));
    hipMemcpy(gpu_btw,g1->btw,g1->N*sizeof(double),hipMemcpyHostToDevice);
    struct cuda_copies cpy1 = copy_to_gpu( g1 );
    for(int i=0;i<aff_comp[0].size();i++)
    {
        up_comp(g1,aff_comp[0][i],cpy1,-1,gpu_btw);
    }
    struct cuda_copies cpy2 = copy_to_gpu( g2 );
    for(int i=0;i<aff_comp[1].size();i++)
    {
        up_comp(g2,aff_comp[1][i],cpy2,1,gpu_btw);
    }
    g2->btw = (double *)malloc((g2->N)*sizeof(double));
    hipMemcpy(g2->btw,gpu_btw,(g2->N)*sizeof(double),hipMemcpyDeviceToHost);
    cout<<"updated\n";
    for(int i=0;i<g2->N;i++)
    {
        cout<<g2->btw[i]<<" ";   
    }
    cout<<"\n";
    *g=g2;
}

__device__ void  init_vec(struct vec* v)
{
    v->size=0;
    v->max_size=4;
    v->arr = (int*) malloc (4*sizeof(int));
}

__device__ void  push(struct vec *v1,int a)
{
    int s_max = v1->max_size;
    int* arr = v1->arr;
    int s = v1->size;
    if(s<s_max)
    {
        arr[s]=a;
        v1->size++;
    }
    else 
    {
        int* arr2 = (int*) malloc (2*s_max*sizeof(int));
        for(int i=0;i<s;i++)
        {
            arr2[i]=arr[i];
        }
        free(arr);
        arr2[s]=a;
        v1->arr = arr2;
        v1->size++;
        v1->max_size = 2*s_max;
    }
}

